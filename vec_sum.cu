#include "hip/hip_runtime.h"
#include "stdio.h"
#include<math.h>
__global__ void addkernel(int *a,int *b,int *c,int n ){

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<n){
		c[i] = a[i] + b[i];
	}

}

void add(int *a,int *b,int *c,int n){

	int *ca,*cb ,*cc ;
	int size = n*sizeof(int);
	hipMalloc((void **)&ca ,size );
	hipMemcpy(ca,a,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&cb ,size );
	hipMemcpy(cb,b,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&cc,size);

	addkernel<<<ceil(n/256.0),256>>>(ca,cb,cc,n);
	hipMemcpy(c,cc,size,hipMemcpyDeviceToHost);

}

int main(){
	clock_t t;
	int k = 100000;
	int a[k],b[k],c[k];
	for(int i=0;i<k;i++){
		a[i] = i;
		b[i] = i;	
	}


	t = clock();
	add(a,b,c,k);
	t = clock() - t;
	double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
    printf("\n%f", time_taken);
	return 0;
}